
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <cmath>

/**
 * CUDA kernel matrix multiplication
*/
__global__ void matrixMulKernel(int* d_A, int* d_B, int* d_C, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N){
        int sum = 0;
        for(int i = 0; i < N; i++){
            sum += d_A[row * N + i] * d_B[i * N + col];
        }
        d_C[row * N + col] = sum;
    }
}


template<typename T>
__global__ void naive_matrix_multiply(const T *A, const T *B, T* C, int width, int P, int Q)
{
  int r = blockIdx.y * blockDim.y + threadIdx.y;   
  int c = blockIdx.x * blockDim.x + threadIdx.x;
  // check boundry conditions
  if( r < P && c < Q){
    // do the multiplication for one row and col
    T value = 0;
    for(int k = 0; k < width; k++){
      value += A[r * width + k] * B[k * Q + c];
    }
    // store the result
    C[r * Q + c] = value;
  }}

int main(void){
    // Matrix size
    int N = 4;
    size_t bytes = N * N * sizeof(int);

    // Host memory
    int* h_A, *h_B, *h_C;
    h_A = (int*)malloc(bytes);
    h_B = (int*)malloc(bytes);
    h_C = (int*)malloc(bytes);

    // Device memory
    int* d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Initialize matrices
    for(int i = 0; i < N * N; i++){
        h_A[i] = 1;
        h_B[i] = i;
    }

    // Print matrices
    printf("Matrix A\n");
    for (size_t i = 0; i < N; i++)
    {
        for (size_t j = 0; j < N; j++)
        {
            printf("%d ", h_A[i * N + j]);
        }
        printf("\n");
    }
    
    printf("\n\n");
    printf("Matrix B\n");
    for(size_t i = 0; i < N; i++){
        for (size_t j = 0; j < N; j++)
        {
            printf("%d ", h_B[i * N + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // Transfer data to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Kernel launch
    dim3 threads(2, 2);
    dim3 blocks(N / threads.x, N / threads.y);
    matrixMulKernel<<<blocks, threads>>>(d_A, d_B, d_C, N);

    // Transfer data back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Print result
    printf("Matrix C\n");
    for(size_t i = 0; i < N; i++){
        for (size_t j = 0; j < N; j++)
        {
            printf("%d ", h_C[i * N + j]);
        }
        printf("\n");
    }
    printf("\n\n");


    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;

}