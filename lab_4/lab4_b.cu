
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <cmath>

/**
 * CUDA kernel matrix multiplication
*/
__global__ void matrixMulKernel(int* d_A, int* d_B, int* d_C, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N){
        int sum = 0;
        for(int i = 0; i < N; i++){
            sum += d_A[row * N + i] * d_B[i * N + col];
        }
        d_C[row * N + col] = sum;
    }
}


int main(void){
    // Open CSV file to write timing results
    std::ofstream file("timing_results.csv");
    if (!file.is_open()) {
        std::cerr << "Error: Unable to open the file." << std::endl;
        return 1;
    }

    // Write CSV header
    file << "Matrix_Size, Execution_Time" << std::endl;

    // Matrix sizes to test
    int sizes[] = {2, 4, 8, 16, 32, 64, 128, 256, 512};

    // Number of runs for averaging
    const int num_runs = 100;

    // Loop through each matrix size
    for (int size : sizes) {
        // Matrix size
        int N = size;
        size_t bytes = N * N * sizeof(int);

        // Timing accumulator
        double total_duration = 0.0;

        // Loop for averaging
        for (int run = 0; run < num_runs; ++run) {
            // Host memory
            int* h_A, *h_B, *h_C;
            h_A = (int*)malloc(bytes);
            h_B = (int*)malloc(bytes);
            h_C = (int*)malloc(bytes);

            // Device memory
            int* d_A, *d_B, *d_C;
            hipMalloc(&d_A, bytes);
            hipMalloc(&d_B, bytes);
            hipMalloc(&d_C, bytes);

            // Initialize matrices
            for(int i = 0; i < N * N; i++){
                h_A[i] = 1;
                h_B[i] = i;
            }

            // Transfer data to device
            hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
            hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

            // Kernel launch
            dim3 threads(2, 2);
            dim3 blocks(N / threads.x, N / threads.y);

            auto start = std::chrono::high_resolution_clock::now();
            matrixMulKernel<<<blocks, threads>>>(d_A, d_B, d_C, N);
            hipDeviceSynchronize();
            auto end = std::chrono::high_resolution_clock::now();

            // Calculate execution time in microseconds
            auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

            // Accumulate timing results
            total_duration += duration;

            // Free memory
            free(h_A);
            free(h_B);
            free(h_C);
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);
        }

        // Calculate average timing results
        double average_duration = total_duration / num_runs;

        // Write average timing results to CSV file
        file << N << ", " << average_duration << std::endl;
    }

    // Close the CSV file
    file.close();

    return 0;
}
