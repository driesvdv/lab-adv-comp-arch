
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <cmath>

// #define M 512       // Lenna width
// #define N 512       // Lenna height
#define M 941     // VR width
#define N 704     // VR height
#define C 3       // Colors
#define OFFSET 15 // Header length

uint8_t *get_image_array(void)
{
    /*
     * Get the data of an (RGB) image as a 1D array.
     *
     * Returns: Flattened image array.
     *
     * Noets:
     *  - Images data is flattened per color, column, row.
     *  - The first 3 data elements are the RGB components
     *  - The first 3*M data elements represent the firts row of the image
     *  - For example, r_{0,0}, g_{0,0}, b_{0,0}, ..., b_{0,M}, r_{1,0}, ..., b_{b,M}, ..., b_{N,M}
     *
     */
    // Try opening the file
    FILE *imageFile;
    imageFile = fopen("./images/input_image.ppm", "rb");
    if (imageFile == NULL)
    {
        perror("ERROR: Cannot open input file");
        exit(EXIT_FAILURE);
    }

    // Initialize empty image array
    uint8_t *image_array = (uint8_t *)malloc(M * N * C * sizeof(uint8_t) + OFFSET);

    // Read the image
    fread(image_array, sizeof(uint8_t), M * N * C * sizeof(uint8_t) + OFFSET, imageFile);

    // Close the file
    fclose(imageFile);

    // Move the starting pointer and return the flattened image array
    return image_array + OFFSET;
}

void save_image_array(uint8_t *image_array)
{
    /*
     * Save the data of an (RGB) image as a pixel map.
     *
     * Parameters:
     *  - param1: The data of an (RGB) image as a 1D array
     *
     */
    // Try opening the file
    FILE *imageFile;
    imageFile = fopen("./output_image.ppm", "wb");
    if (imageFile == NULL)
    {
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }

    // Configure the file
    fprintf(imageFile, "P6\n");          // P6 filetype
    fprintf(imageFile, "%d %d\n", M, N); // dimensions
    fprintf(imageFile, "255\n");         // Max pixel

    // Write the image
    fwrite(image_array, 1, M * N * C, imageFile);

    // Close the file
    fclose(imageFile);
}

/**
 * CUDA Kernel Device code
 *
 * Makes the image grayscale by using the average RGB method using coalesced memory access
 */
__global__ void coalesced_memory_acces(uint8_t *image, int numPixels)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numPixels)
    {
        // calculate the average grayscale value
        int gray = (image[idx * C] + image[idx * C + 1] + image[idx * C + 2]) / 3;

        // set the rgb pixels to the newly calculated value
        image[idx * C] = gray;
        image[idx * C + 1] = gray;
        image[idx * C + 2] = gray;
    }
}

/**
 * CUDA Kernel Device code
 *
 * Makes the image grayscale by using the average RGB method using non coalesced memory access
 */
__global__ void non_coalesced_memory_acces(uint8_t *image, int numPixels, int channel)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numPixels)
    {
        // calculate the average grayscale value
        int gray = (image[idx * C] + image[idx * C + 1] + image[idx * C + 2]) / 3;

        // set the rgb pixels to the newly calculated value
        image[idx * C] = gray;
    }
}



int main(void)
{
    // Open CSV file for writing
    std::ofstream file("timing_data_divergence_fix.csv");
    file << "Threads, Execution Time (µs)\n";

    // Read the image
    uint8_t *h_image_array = get_image_array();

    // Calculate total number of pixels
    int numPixels = M * N;

    // Allocate memory on the GPU for the image
    uint8_t *d_image_array;

    // Allocate memory on the GPU for the image
    hipMalloc((void **)&d_image_array, numPixels * C * sizeof(uint8_t));

    // Copy the image data from host to device
    hipMemcpy(d_image_array, h_image_array, numPixels * C * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    int blockSize = 512;
    int numBlocks = ceil((double)(numPixels) / blockSize);
    // int warps = numStrides * ceil((double)(blockSize) / 32);

    auto start = std::chrono::high_resolution_clock::now();

    // Launch the kernel to grayscale image using coalesced memory access
    coalesced_memory_acces<<<numBlocks, blockSize>>>(d_image_array, numPixels);

    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::micro> duration = end - start;
    printf("Coalesced execution Time: %f µs\n", duration.count());

    // Copy the inverted image data back to host
    hipMemcpy(h_image_array, d_image_array, numPixels * C * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // Print output array
    for (int i = 0; i < 100; i++)
    {
        printf("%d ", h_image_array[i]);
    }

    // Save the output image
    save_image_array(h_image_array);

    // Free device memory
    hipFree(d_image_array);

    // Close CSV file
    file.close();

    printf("Done\n");
    return 0;
}

// order the rgb values in array like this: r0, r1, r2, g1, g2, g3, b1, b2, b3,...

